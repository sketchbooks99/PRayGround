#include "hip/hip_runtime.h"
#include "util.cuh"
#include <prayground/texture/bitmap.h>
#include <prayground/texture/constant.h>
#include <prayground/texture/checker.h>

using namespace prayground;

extern "C" __device__ float3 __direct_callable__bitmap(SurfaceInteraction* si, void* tex_data) {
    const BitmapTextureData* image = reinterpret_cast<BitmapTextureData*>(tex_data);
    float4 c = tex2D<float4>(image->texture, si->uv.x, si->uv.y);
    return make_float3(c.x, c.y, c.z);
}

extern "C" __device__ float3 __direct_callable__constant(SurfaceInteraction* si, void* tex_data) {
    const ConstantTextureData* constant = reinterpret_cast<ConstantTextureData*>(tex_data);
    return make_float3(constant->color);
}

extern "C" __device__ float3 __direct_callable__checker(SurfaceInteraction* si, void* tex_data) {
    const CheckerTextureData* checker = reinterpret_cast<CheckerTextureData*>(tex_data);
    const bool is_odd = sinf(si->uv.x*math::pi*checker->scale) * sinf(si->uv.y*math::pi*checker->scale) < 0;
    return is_odd ? make_float3(checker->color1) : make_float3(checker->color2);
}