#include "hip/hip_runtime.h"
#include "util.cuh"

extern "C" __device__ Vec3f __direct_callable__bitmap(SurfaceInteraction* si, void* tex_data) {
    const auto* image = reinterpret_cast<BitmapTexture::Data*>(tex_data);
    float4 c = tex2D<float4>(image->texture, si->shading.uv.x(), si->shading.uv.y());
    return Vec3f(c);
}

extern "C" __device__ Vec3f __direct_callable__constant(SurfaceInteraction* si, void* tex_data) {
    const auto* constant = reinterpret_cast<ConstantTexture::Data*>(tex_data);
    return constant->color;
}

extern "C" __device__ Vec3f __direct_callable__checker(SurfaceInteraction* si, void* tex_data) {
    const auto* checker = reinterpret_cast<CheckerTexture::Data*>(tex_data);
    const bool is_odd = sinf(si->shading.uv.x() * math::pi * checker->scale) * sinf(si->shading.uv.y() * math::pi * checker->scale) < 0;
    return lerp(checker->color1, checker->color2, (float)is_odd);
}