#include "hip/hip_runtime.h"
#include <prayground/prayground.h>
#include "params.h"

using SurfaceInteraction = SurfaceInteraction_<Vec3f>;

extern "C" { __constant__ LaunchParams params; }

INLINE DEVICE SurfaceInteraction* getSurfaceInteraction()
{
    const uint32_t u0 = optixGetPayload_0();
    const uint32_t u1 = optixGetPayload_1();
    return reinterpret_cast<SurfaceInteraction*>( unpackPointer(u0, u1) ); 
}

// -------------------------------------------------------------------------------
INLINE DEVICE void trace(
    OptixTraversableHandle handle, const Vec3f& ro, const Vec3f& rd,
    float tmin, float tmax, uint32_t ray_type, SurfaceInteraction* si) 
{
    uint32_t u0, u1;
    packPointer( si, u0, u1 );
    optixTrace(
        handle, ro, rd,
        tmin, tmax, 0.0f,
        OptixVisibilityMask( 1 ),
        OPTIX_RAY_FLAG_NONE,
        ray_type, 1, ray_type,        
        u0, u1 );	
}

static __forceinline__ __device__ Vec3f reinhardToneMap(const Vec3f& color, const float white)
{
    const float l = luminance(color);
    return (color * 1.0f) / (1.0f + l / white);
}

extern "C" __device__ void __raygen__pinhole()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const int frame = params.frame;
    const Vec3ui idx(optixGetLaunchIndex());
    uint32_t seed = tea<4>(idx.y() * params.width + idx.x(), frame);

    Vec3f result(0.0f);

    int i = params.samples_per_launch;

    do
    {
        const Vec2f jitter = UniformSampler::get2D(seed) - 0.5f;
        const Vec2f res(params.width, params.height);
        const Vec2f d = 2.0f * ((Vec2f(idx.x(), idx.y()) + jitter) / res) - 1.0f;

        Vec3f ro, rd;
        getCameraRay(raygen->camera, d.x(), d.y(), ro, rd);

        Vec3f throughput(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = Vec3f(0.0f);
        si.albedo = Vec3f(0.0f);
        si.trace_terminate = false;

        float tmax = raygen->camera.farclip / dot(rd, normalize(raygen->camera.lookat - ro));

        int depth = 0;
        for ( ;; ) {

            if ( depth >= params.max_depth )
				break;

            trace(params.handle, ro, rd, 0.01f, tmax, 0, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            // Get emission from area emitter
            if ( si.surface_info.type == SurfaceType::AreaEmitter )
            {
                // Evaluating emission from emitter
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                result += si.emission * throughput;

                if (si.trace_terminate)
                    break;
            }
            // Specular sampling
            else if (+(si.surface_info.type & SurfaceType::Delta))
            {
                // Sampling scattered direction
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);
                
                // Evaluate bsdf
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                throughput *= bsdf_val;
            }
            // Rough surface sampling with applying MIS
            else if ( +(si.surface_info.type & (SurfaceType::Rough | SurfaceType::Diffuse)) )
            {
                // Importance sampling according to the BSDF
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);

                // Evaluate PDF depends on BSDF
                float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.pdf, &si, si.surface_info.data);

                // Evaluate BSDF
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);

                throughput *= bsdf_val / bsdf_pdf;
            }

            // Make tmax large except for when the primary ray
            tmax = 1e16f;
            
            ro = si.p;
            rd = si.wi;

            ++depth;
        }
    } while (--i);

    const uint32_t image_index = idx.y() * params.width + idx.x();

    if (result.x() != result.x()) result.x() = 0.0f;
    if (result.y() != result.y()) result.y() = 0.0f;
    if (result.z() != result.z()) result.z() = 0.0f;

    Vec3f accum_color = result / static_cast<float>(params.samples_per_launch);

    if (frame > 0)
    {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const Vec3f accum_color_prev(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = Vec4f(accum_color, 1.0f);
    Vec3u color = make_color(reinhardToneMap(accum_color, params.white));
    params.result_buffer[image_index] = Vec4u(color, 255);
}

// Triangle mesh -------------------------------------------------------------------------------
extern "C" __device__ void __miss__envmap()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    const auto* env = reinterpret_cast<EnvironmentEmitter::Data*>(data->env_data);
    SurfaceInteraction* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f*1e8f;
    const float discriminant = half_b * half_b - a*c;

    float sqrtd = sqrtf(discriminant);
    float t = (-half_b + sqrtd) / a;

    Vec3f p = normalize(ray.at(t));

    float phi = atan2(p.z(), p.x());
    float theta = asin(p.y());
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) * math::inv_pi;
    si->shading.uv = make_float2(u, v);
    si->trace_terminate = true;
    si->surface_info.type = SurfaceType::None;
    si->emission = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        env->texture.prg_id, si, env->texture.data);
}

// Triangle mesh -------------------------------------------------------------------------------
extern "C" __device__ void __closesthit__mesh()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const auto* mesh = reinterpret_cast<TriangleMesh::Data*>(data->shape_data);

    Ray ray = getWorldRay();
    
    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh->faces[prim_id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const Vec2f texcoord0 = mesh->texcoords[face.texcoord_id.x()];
    const Vec2f texcoord1 = mesh->texcoords[face.texcoord_id.y()];
    const Vec2f texcoord2 = mesh->texcoords[face.texcoord_id.z()];
    const Vec2f texcoords = (1-u-v)*texcoord0 + u*texcoord1 + v*texcoord2;

    const Vec3f n0 = mesh->normals[face.normal_id.x()];
	const Vec3f n1 = mesh->normals[face.normal_id.y()];
	const Vec3f n2 = mesh->normals[face.normal_id.z()];

    // Linear interpolation of normal by barycentric coordinates.
    Vec3f local_n = (1.0f-u-v)*n0 + u*n1 + v*n2;
    Vec3f world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wo = ray.d;
    si->shading.uv = texcoords;
    si->surface_info = data->surface_info;
}

// Surface functions ------------------------------------------------------------------------------------------
// Diffuse -----------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_diffuse(SurfaceInteraction* si, void* mat_data) {
    const auto* diffuse = reinterpret_cast<Diffuse::Data*>(mat_data);

    if (diffuse->twosided)
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);

    si->trace_terminate = false;
    uint32_t seed = si->seed;
    Vec2f u = UniformSampler::get2D(seed);
    Vec3f wi = cosineSampleHemisphere(u[0], u[1]);
    Onb onb(si->shading.n);
    onb.inverseTransform(wi);
    si->wi = normalize(wi);
    si->seed = seed;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_diffuse(SurfaceInteraction* si, void* mat_data)
{
    const auto* diffuse = reinterpret_cast<Diffuse::Data*>(mat_data);
    const Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        diffuse->texture.prg_id, si, diffuse->texture.data);
    si->albedo = albedo;
    si->emission = Vec3f(0.0f);
    const float cosine = fmaxf(0.0f, dot(si->shading.n, si->wi));
    return albedo * cosine * math::inv_pi;
}

extern "C" __device__ float __direct_callable__pdf_diffuse(SurfaceInteraction* si, void* mat_data)
{
    const float cosine = fmaxf(0.0f, dot(si->shading.n, si->wi));
    return cosine * math::inv_pi;
}

// Dielectric --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_dielectric(SurfaceInteraction* si, void* mat_data) {
    const auto* dielectric = reinterpret_cast<Dielectric::Data*>(mat_data);

    float ni = 1.000292f; // air
    float nt = dielectric->ior;  // ior specified 
    float cosine = dot(si->wo, si->shading.n);
    bool into = cosine < 0;
    Vec3f outward_normal = into ? si->shading.n : -si->shading.n;

    if (!into) swap(ni, nt);

    cosine = fabs(cosine);
    float sine = sqrtf(1.0 - cosine*cosine);
    bool cannot_refract = (ni / nt) * sine > 1.0f;

    float reflect_prob = fresnel(cosine, ni, nt);
    unsigned int seed = si->seed;

    if (cannot_refract || reflect_prob > rnd(seed))
        si->wi = reflect(si->wo, outward_normal);
    else    
        si->wi = refract(si->wo, outward_normal, cosine, ni, nt);
    si->trace_terminate = false;
    si->seed = seed;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_dielectric(SurfaceInteraction* si, void* mat_data)
{
    const auto* dielectric = reinterpret_cast<Dielectric::Data*>(mat_data);
    si->emission = Vec3f(0.0f);
    Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        dielectric->texture.prg_id, si, dielectric->texture.data);
    si->albedo = albedo;
    return albedo;
}

extern "C" __device__ float __direct_callable__pdf_dielectric(SurfaceInteraction* si, void* mat_data)
{
    return 1.0f;
}

// Conductor --------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__sample_conductor(SurfaceInteraction* si, void* mat_data) {
    const auto* conductor = reinterpret_cast<Conductor::Data*>(mat_data);
    if (conductor->twosided)
        si->shading.n = faceforward(si->shading.n, -si->wo, si->shading.n);

    si->wi = reflect(si->wo, si->shading.n);
    si->trace_terminate = false;
}

extern "C" __device__ Vec3f __continuation_callable__bsdf_conductor(SurfaceInteraction* si, void* mat_data)
{
    const auto* conductor = reinterpret_cast<Conductor::Data*>(mat_data);
    si->emission = Vec3f(0.0f);
    Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        conductor->texture.prg_id, si, conductor->texture.data);
    si->albedo = albedo;
    return albedo;
}

extern "C" __device__ float __direct_callable__pdf_conductor(SurfaceInteraction* si, void* mat_data)
{
    return 1.0f;
}

// Area emitter ------------------------------------------------------------------------------------------
extern "C" __device__ void __direct_callable__area_emitter(SurfaceInteraction* si, void* surface_data)
{
    const auto* area = reinterpret_cast<AreaEmitter::Data*>(surface_data);
    si->trace_terminate = true;
    float is_emitted = dot(si->wo, si->shading.n) < 0.0f ? 1.0f : 0.0f;
    if (area->twosided)
    {
        is_emitted = 1.0f;
        si->shading.n = faceforward(si->shading.n, -si->wi, si->shading.n);
    }

    const Vec3f base = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        area->texture.prg_id, si, area->texture.data);
    si->albedo = base;
    
    si->emission = base * area->intensity * is_emitted;
}

// Texture functions -------------------------------------------------------------------------------
extern "C" __device__ Vec3f __direct_callable__bitmap(SurfaceInteraction* si, void* tex_data) {
    const auto* image = reinterpret_cast<BitmapTexture::Data*>(tex_data);
    float4 c = tex2D<float4>(image->texture, si->shading.uv.x(), si->shading.uv.y());
    return Vec3f(c);
}

extern "C" __device__ Vec3f __direct_callable__constant(SurfaceInteraction* si, void* tex_data) {
    const auto* constant = reinterpret_cast<ConstantTexture::Data*>(tex_data);
    return constant->color;
}

extern "C" __device__ Vec3f __direct_callable__checker(SurfaceInteraction* si, void* tex_data) {
    const auto* checker = reinterpret_cast<CheckerTexture::Data*>(tex_data);
    const bool is_odd = sinf(si->shading.uv.x() * math::pi * checker->scale) * sinf(si->shading.uv.y() * math::pi * checker->scale) < 0;
    return lerp(checker->color1, checker->color2, (float)is_odd);
}