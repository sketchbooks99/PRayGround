#include "hip/hip_runtime.h"
#include "util.cuh"
#include <prayground/emitter/envmap.h>
#include <prayground/core/ray.h>

using namespace prayground;

extern "C" __device__ void __miss__envmap()
{
    MissData* data = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
    EnvironmentEmitterData* env = reinterpret_cast<EnvironmentEmitterData*>(data->env_data);
    SurfaceInteraction* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f*1e8f;
    const float discriminant = half_b * half_b - a*c;

    float sqrtd = sqrtf(discriminant);
    float t = (-half_b + sqrtd) / a;

    float3 p = normalize(ray.at(t));

    float phi = atan2(p.z, p.x);
    float theta = asin(p.y);
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    
    si->uv = make_float2(u, v);
    si->n = make_float3(0.0f);
    si->p = p;
    float3 color = optixDirectCall<float3, SurfaceInteraction*, void*>(
        env->tex_program_id, si, env->tex_data
    );
    si->shading_val = color;
    si->albedo = color;
}