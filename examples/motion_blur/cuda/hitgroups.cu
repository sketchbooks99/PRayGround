#include "hip/hip_runtime.h"
#include "util.cuh"

using namespace prayground;

extern "C" __device__ void __intersection__plane()
{
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const PlaneData* plane_data = reinterpret_cast<PlaneData*>(data->shape_data);

    const float2 min = plane_data->min;
    const float2 max = plane_data->max;

    Ray ray = getLocalRay();

    const float t = -ray.o.y / ray.d.y;

    const float x = ray.o.x + t * ray.d.x;
    const float z = ray.o.z + t * ray.d.z;

    float2 uv = make_float2((x - min.x) / (max.x - min.x), (z - min.y) / (max.y - min.y));

    if (min.x < x && x < max.x && min.y < z && z < max.y && ray.tmin < t && t < ray.tmax)
        optixReportIntersection(t, 0, float2_as_ints(uv));
}

extern "C" __device__ void __closesthit__plane()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());

    Ray ray = getWorldRay();

    float3 local_n = make_float3(0, 1, 0);
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);
    float2 uv = make_float2(
        int_as_float( optixGetAttribute_0() ), 
        int_as_float( optixGetAttribute_1() )
    );

    SurfaceInteraction* si = getSurfaceInteraction();

    si->p = ray.at(ray.tmax);
    si->n = faceforward(world_n, -ray.d, world_n);
    si->uv = uv;
    float3 albedo = optixDirectCall<float3, SurfaceInteraction*, void*>(
        data->tex_data.prg_id, si, data->tex_data.data
    );
    si->albedo = albedo;

    const float3 light_dir = normalize(params.light.pos - si->p);
    si->shading_val = 0.8f * fmaxf(0.0f, dot(light_dir, si->n)) * albedo + 0.2f * albedo;
}

extern "C" __device__ void __closesthit__mesh()
{
    HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const MeshData* mesh_data = reinterpret_cast<MeshData*>(data->shape_data);

    Ray ray = getWorldRay();
    
    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh_data->faces[prim_id];
    const float u = optixGetTriangleBarycentrics().x;
    const float v = optixGetTriangleBarycentrics().y;

    const float2 texcoord0 = mesh_data->texcoords[face.texcoord_id.x];
    const float2 texcoord1 = mesh_data->texcoords[face.texcoord_id.y];
    const float2 texcoord2 = mesh_data->texcoords[face.texcoord_id.z];
    const float2 texcoords = (1-u-v)*texcoord0 + u*texcoord1 + v*texcoord2;

    float3 n0 = normalize(mesh_data->normals[face.normal_id.x]);
	float3 n1 = normalize(mesh_data->normals[face.normal_id.y]);
	float3 n2 = normalize(mesh_data->normals[face.normal_id.z]);

    // Linear interpolation of normal by barycentric coordinates.
    float3 local_n = (1.0f-u-v)*n0 + u*n1 + v*n2;
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = faceforward(world_n, -ray.d, world_n);
    si->uv = texcoords;
    float3 albedo = optixDirectCall<float3, SurfaceInteraction*, void*>(
        data->tex_data.prg_id, si, data->tex_data.data
    );
    si->albedo = albedo;

    const float3 light_dir = normalize(params.light.pos - si->p);
    si->shading_val = 0.8f * fmaxf(0.0f, dot(light_dir, si->n)) * albedo + 0.2f * albedo;
}

static __forceinline__ __device__ float2 getUV(const float3& p) {
    float phi = atan2(p.z, p.x);
    float theta = asin(p.y);
    float u = 1.0f - (phi + math::pi) / (2.0f * math::pi);
    float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    return make_float2(u, v);
}

extern "C" __device__ void __intersection__sphere() {
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    const float3 center = sphere_data->center;
    const float radius = sphere_data->radius;

    Ray ray = getLocalRay();

    const float3 oc = ray.o - center;
    const float a = dot(ray.d, ray.d);
    const float half_b = dot(oc, ray.d);
    const float c = dot(oc, oc) - radius * radius;
    const float discriminant = half_b * half_b - a * c;

    if (discriminant > 0.0f) {
        float sqrtd = sqrtf(discriminant);
        float t1 = (-half_b - sqrtd) / a;
        bool check_second = true;
        if (t1 > ray.tmin && t1 < ray.tmax) {
            float3 normal = normalize((ray.at(t1) - center) / radius);
            check_second = false;
            optixReportIntersection(t1, 0, float3_as_ints(normal));
        }

        if (check_second) {
            float t2 = (-half_b + sqrtd) / a;
            if (t2 > ray.tmin && t2 < ray.tmax) {
                float3 normal = normalize((ray.at(t2) - center) / radius);
                optixReportIntersection(t2, 0, float3_as_ints(normal));
            }
        }
    }
}

extern "C" __device__ void __closesthit__sphere() {
    const HitgroupData* data = reinterpret_cast<HitgroupData*>(optixGetSbtDataPointer());
    const SphereData* sphere_data = reinterpret_cast<SphereData*>(data->shape_data);

    Ray ray = getWorldRay();

    float3 local_n = make_float3(
        int_as_float(optixGetAttribute_0()),
        int_as_float(optixGetAttribute_1()),
        int_as_float(optixGetAttribute_2())
    );
    float3 world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    SurfaceInteraction* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->n = faceforward(world_n, -ray.d, world_n);
    si->uv = getUV(local_n);
    float3 albedo = optixDirectCall<float3, SurfaceInteraction*, void*>(
        data->tex_data.prg_id, si, data->tex_data.data
    );
    si->albedo = albedo;
    const float3 light_dir = normalize(params.light.pos - si->p);
    si->shading_val = 0.8f * fmaxf(0.0f, dot(light_dir, si->n)) * albedo + 0.2f * albedo;
}
