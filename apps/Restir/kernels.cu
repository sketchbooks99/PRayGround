#include "hip/hip_runtime.h"
#include <prayground/prayground.h>
#include "params.h"

extern "C" { __constant__ LaunchParams params; }

using SurfaceInteraction = SurfaceInteraction_<Vec3f>;

struct Reservoir {
    int y;          // The output sample (the index of light)
    float wsum;     // The sum of weights
    int M;          // The number of samples seen so far
    float W;        // Probalistic weight

    void update(int i, float weight, uint32_t& seed)
    {
        wsum += weight;
        M++;
        if (rnd(seed) < (weight / wsum))
            y = i;
    }
};

static __forceinline__ __device__ float calcWeight(const Vec3f& sample)
{
    /// @todo provisional value
    return 1.0f;
}

static __forceinline__ __device__ Reservoir reservoirSampling(int32_t num_strategies, Vec3f sample, int* samples, uint32_t& seed)
{
    Reservoir r;
    for (int i = 0; i < num_strategies; i++)
        r.update(samples[i], calcWeight(sample), seed);
    return r;
}

// p^(x) = \rho(x) * Le(x) * G(x), where \rho(x) = BRDF
static __forceinline__ __device__ float targetPDF(
    const Vec3f& brdf, SurfaceInteraction* si, const Vec3f& to_light, const LightInfo& light)
{
    float3 N = light.triangle.n;
    N = faceforward(N, normalize(-to_light), N);
    const float area = length(cross(light.triangle.v1 - light.triangle.v0, light.triangle.v2 - light.triangle.v0)) * 0.5f;
    const float cos_theta = fmaxf(dot(light.triangle.n, normalize(-to_light)), 0.0001f);
    const float d = length(to_light);
    const float G = (d * d) / (area * cos_theta);

    return length(brdf * light.emission) * G;
}

static __forceinline__ __device__ Vec3f randomSampleOnTriangle(uint32_t& seed, const Triangle& triangle)
{
    // Uniform sampling of barycentric coordinates on a triangle
    Vec2f uv = UniformSampler::get2D(seed);
    return triangle.v0 * (1.0f - uv.x() - uv.y()) + triangle.v1 * uv.x() + triangle.v2 * uv.y();
}

static __forceinline__ __device__ Reservoir reservoirImportanceSampling(
    SurfaceInteraction* si, int M, uint32_t& seed)
{
    Reservoir r{ 0, 0, 0, 0 };
    for (int i = 0; i < min(params.num_lights, M); i++)
    {
        // Sample a light
        int light_idx = rndInt(seed, 0, params.num_lights - 1);
        LightInfo light = params.lights[light_idx];

        const Vec3f light_p = randomSampleOnTriangle(seed, light.triangle);

        // Get brdf wrt the sampled light
        Vec3f brdf = optixContinuationCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
            si->surface_info.callable_id.bsdf, si, si->surface_info.data, light_p);
        float pdf = optixDirectCall<float, SurfaceInteraction*, void*, const Vec3f&>(
            si->surface_info.callable_id.pdf, si, si->surface_info.data, light_p);
        pdf = fmaxf(pdf, 0.001f);
        // Get target pdf 
        const float target_pdf = targetPDF(brdf, si, light_p - si->p, light);
        // update reservoir
        r.update(light_idx, target_pdf / pdf, seed);
    }

    LightInfo light = params.lights[r.y];
    const Vec3f light_p = randomSampleOnTriangle(seed, light.triangle);
    Vec3f brdf = optixContinuationCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
        si->surface_info.callable_id.bsdf, si, si->surface_info.data, light_p);

    // r.W = ( ( 1.0f / targetPDF(brdf, si, light_p - si->p, light) ) * ( 1.0f / r.M ) * r.wsum );
    //r.W = fmaxf(( 1.0f / targetPDF(brdf, si, light_p - si->p, light) ) * ( 1.0f / (float)r.M ) * r.wsum, 0.0f);
    r.W = fmaxf( (1.0f / targetPDF(brdf, si, light_p - si->p, light)) * (1.0f / (float)r.M) * r.wsum, 0.0f);
    if (r.W != r.W) 
        r.W = 0.0f;

    //printf("Reservoir: r.y: %d, r.wsum: %f, r.M: %d, r.W: %f\n", r.y, r.wsum, r.M, r.W);
    return r;
}

static __forceinline__ __device__ SurfaceInteraction* getSurfaceInteraction()
{
    const uint32_t u0 = getPayload<0>();
    const uint32_t u1 = getPayload<1>();
    return reinterpret_cast<SurfaceInteraction*>(unpackPointer(u0, u1));
}

static __forceinline__ __device__ void trace(
    OptixTraversableHandle handle, const Vec3f& ro, const Vec3f& rd, 
    float tmin, float tmax, SurfaceInteraction* si)
{
    uint32_t u0, u1;
    packPointer(si, u0, u1);
    optixTrace(
        handle, ro, rd, 
        tmin, tmax, 0, 
        OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 
        (uint32_t)RayType::Radiance, (uint32_t)RayType::NRay, (uint32_t)RayType::Radiance, 
        u0, u1);
}

static __forceinline__ __device__ bool traceShadow(
    OptixTraversableHandle handle, const Vec3f& ro, const Vec3f& rd,
    float tmin, float tmax, SurfaceInteraction* si)
{
    uint32_t hit = 0u;
    optixTrace(
        handle, ro, rd, 
        tmin, tmax, 0.0f,
        OptixVisibilityMask(1), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
        (uint32_t)RayType::Shadow, (uint32_t)RayType::NRay, (uint32_t)RayType::Shadow,
        hit
    );
    return static_cast<bool>(hit);
}

// raygen 
extern "C" __device__ void __raygen__restir()
{
    const auto* raygen = reinterpret_cast<pgRaygenData<Camera>*>(optixGetSbtDataPointer());

    const int frame = params.frame;
    const Vec3ui idx(optixGetLaunchIndex());
    uint32_t seed = tea<4>(idx.x() * params.width + idx.y(), frame);

    Vec3f result(0.0f);

    int spl = params.samples_per_launch;

    const int M = 32;

    for (int i = 0; i < spl; i++) 
    {
        const Vec2f jitter = UniformSampler::get2D(seed) - 0.5f;

        const Vec2f d = 2.0f * Vec2f(
            (static_cast<float>(idx.x()) + jitter.x()) / params.width,
            (static_cast<float>(idx.y()) + jitter.y()) / params.height
        ) - 1.0f;

        Vec3f ro,rd;
        getCameraRay(raygen->camera, d.x(), d.y(), ro, rd);

        Vec3f throughput(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = Vec3f(0.0f);
        si.albedo = Vec3f(0.0f);
        si.trace_terminate = false;

        int depth = 0;
        for (;;) 
        {
            if ( depth >= params.max_depth || si.trace_terminate)
                break;

            trace(params.handle, ro, rd, 0.01f, 1e16f, &si);

            if (si.trace_terminate)
            {
                result += si.emission * throughput;
                break;
            }

            if ( si.surface_info.type == SurfaceType::AreaEmitter )
            {
                // Evaluation of emittance from area emitter
                const Vec3f emission = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                result += emission * throughput;
                if (si.trace_terminate)
                    break;
            }
            else 
            {
                
                Reservoir r = reservoirImportanceSampling(&si, M, seed);
                LightInfo light = params.lights[r.y];
                const Vec3f light_p = randomSampleOnTriangle(seed, light.triangle);
                const Vec3f to_light = light_p - si.p;

                const float nDl = dot(si.shading.n, normalize(to_light));

                Vec3f LN = light.triangle.n;
                LN = faceforward(LN, normalize(to_light), LN);
                const float LnDl = dot(LN, normalize(to_light));
                // rho
                Vec3f brdf = optixContinuationCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data, light_p);

                float weight = 0.0f;
                if (nDl > 0.0f && LnDl > 0.0f)
                {
                    // Visibility term
                    bool occluded = traceShadow(params.handle, si.p, normalize(to_light), 0.01f, length(to_light) - 0.01f, &si);
                    if (!occluded)
                    {
                        // G
                        const float area = length(cross(light.triangle.v1 - light.triangle.v0, light.triangle.v2 - light.triangle.v0)) * 0.5f;
                        //const float cos_theta = fmaxf(dot(light.triangle.n, normalize(to_light)), 0.0f);
                        const float d = length(to_light);
                        const float G = area * nDl / (d * d);
                        weight = nDl * LnDl * area / (d * d);
                    }
                }
                //result += light.emission * brdf * weight;
                //result += r.W * light.emission * brdf * weight;
                result += r.W * light.emission * brdf;

                //printf("r.W: %f, light.emission: %f %f %f, brdf: %f %f %f, weight: %f\n",
                    //r.W, light.emission.x(), light.emission.y(), light.emission.z(), brdf.x(), brdf.y(), brdf.z(), weight);

                // Uniform hemisphere sampling
                si.trace_terminate = true;
                Vec2f u = UniformSampler::get2D(seed);
                Vec3f wi = cosineSampleHemisphere(u[0], u[1]);
                Onb onb(si.shading.n);
                onb.inverseTransform(wi);
                si.wi = normalize(wi);
                si.seed = seed;

                //const Vec3f brdf = optixDirectCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
                //    si.surface_info.callable_id.bsdf, &si, si.surface_info.data, si.p + si.wi);

                //throughput *= brdf;
            }

            ro = si.p;
            rd = si.wi;

            ++depth;
        }
    }

    const uint32_t image_idx = idx.y() * params.width + idx.x();

    // Nan | Inf check
    if (result.x() != result.x()) result.x() = 0.0f;
    if (result.y() != result.y()) result.y() = 0.0f;
    if (result.z() != result.z()) result.z() = 0.0f;

    Vec3f accum = result / static_cast<float>(spl);

    if (frame > 0)
    {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const Vec3f accum_prev = Vec3f(params.accum_buffer[image_idx]);
        accum = lerp(accum_prev, accum, a);
    }

    params.accum_buffer[image_idx] = Vec4f(accum, 1.0f);
    Vec3u color = make_color(accum);
    params.result_buffer[image_idx] = Vec4u(color, 255);
}

// Miss -------------------------------------------------------------
extern "C" __device__ void __miss__envmap()
{
    const auto* data = (pgMissData*)optixGetSbtDataPointer();
    const auto* env = (EnvironmentEmitter::Data*)data->env_data;
    auto* si = getSurfaceInteraction();

    Ray ray = getWorldRay();

    const float a = dot(ray.d, ray.d);
    const float half_b = dot(ray.o, ray.d);
    const float c = dot(ray.o, ray.o) - 1e8f * 1e8f;
    const float D = half_b * half_b - a * c;

    const float sqrtD = sqrtf(D);
    const float t = (-half_b - sqrtD) / a;

    Vec3f p = normalize(ray.at(t));

    const float phi = atan2(p.z(), p.x());
    const float theta = asin(p.y());
    const float u = 1.0f - (phi + math::pi) / (math::two_pi);
    const float v = 1.0f - (theta + math::pi / 2.0f) / math::pi;
    si->shading.uv = Vec2f(u, v);
    si->trace_terminate = true;
    si->surface_info.type = SurfaceType::None;
    si->emission = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        env->texture.prg_id, si, env->texture.data);
}

extern "C" __device__ void __miss__shadow()
{
    setPayload<0>(0);
}

// Hitgroups ---------------------------------------------------------
// Mesh
extern "C" __device__ void __closesthit__mesh()
{
    const auto* data = (pgHitgroupData*)optixGetSbtDataPointer();
    const auto* mesh = (TriangleMesh::Data*)data->shape_data;
    
    Ray ray = getWorldRay();

    const int prim_id = optixGetPrimitiveIndex();
    const Face face = mesh->faces[prim_id];
    const Vec2f uv = optixGetTriangleBarycentrics();

    const Vec3f p0 = mesh->vertices[face.vertex_id.x()];
    const Vec3f p1 = mesh->vertices[face.vertex_id.y()];
    const Vec3f p2 = mesh->vertices[face.vertex_id.z()];

    const Vec2f texcoord0 = mesh->texcoords[face.texcoord_id.x()];
    const Vec2f texcoord1 = mesh->texcoords[face.texcoord_id.y()];
    const Vec2f texcoord2 = mesh->texcoords[face.texcoord_id.z()];
    const Vec2f texcoord = (1.0f - uv.x() - uv.y()) * texcoord0 + uv.x() * texcoord1 + uv.y() * texcoord2;

    const Vec3f n0 = mesh->normals[face.normal_id.x()];
    const Vec3f n1 = mesh->normals[face.normal_id.y()];
    const Vec3f n2 = mesh->normals[face.normal_id.z()];

    Vec3f local_n = (1.0f - uv.x() - uv.y()) * n0 + uv.x() * n1 + uv.y() * n2;
    Vec3f world_n = optixTransformNormalFromObjectToWorldSpace(local_n);
    world_n = normalize(world_n);

    auto* si = getSurfaceInteraction();
    si->p = ray.at(ray.tmax);
    si->shading.n = world_n;
    si->t = ray.tmax;
    si->wo = ray.d;
    si->shading.uv = texcoord;
    si->surface_info = data->surface_info;

    Vec3f dpdu, dpdv;
    const Vec2f duv02 = texcoord0 - texcoord2, duv12 = texcoord1 - texcoord2;
    const Vec3f dp02 = p0 - p2, dp12 = p1 - p2;
    const float D = duv02.x() * duv12.y() - duv02.y() * duv12.x();
    bool degenerateUV = fabs(D) < 1e-8f;
    if (!degenerateUV)
    {
        const float invD = 1.0f / D;
        dpdu = (duv12.y() * dp02 - duv02.y() * dp12) * invD;
        dpdv = (duv02.x() * dp12 - duv12.x() * dp02) * invD;
    }
    if (degenerateUV || length(cross(dpdu, dpdv)) == 0.0f)
    {
        Onb onb(si->shading.n);
        dpdu = onb.tangent;
        dpdv = onb.bitangent;  
    }
    si->shading.dpdu = normalize(optixTransformNormalFromObjectToWorldSpace(dpdu));
    si->shading.dpdv = normalize(optixTransformNormalFromObjectToWorldSpace(dpdv));
}

extern "C" __device__ void __closesthit__shadow()
{
    setPayload<0>(1);
}

// Surfaces -----------------------------------------------------------------------
// Diffuse
extern "C" __device__ void __direct_callable__sample_diffuse(SurfaceInteraction* si, void* mat_data)
{

}

extern "C" __device__ Vec3f __continuation_callable__brdf_diffuse(SurfaceInteraction * si, void* mat_data, const Vec3f & p)
{
    const auto* diffuse = (Diffuse::Data*)mat_data;
    si->emission = Vec3f(0.0f);

    const Vec3f wi = normalize(p - si->p);
    const Vec3f albedo = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        diffuse->texture.prg_id, si, diffuse->texture.data);
    si->albedo = albedo;
    const float cos_theta = fmaxf(dot(si->shading.n, wi), 0.0f);
    return albedo * cos_theta * math::inv_pi;
}

extern "C" __device__ float __direct_callable__pdf_diffuse(SurfaceInteraction * si, void* mat_data, const Vec3f& p)
{
    const Vec3f wi = normalize(p - si->p);
    const float cos_theta = fmaxf(dot(si->shading.n, wi), 0.0f);
    return cos_theta * math::inv_pi;
}

// Disney
extern "C" __device__ void __direct_callable__sample_disney(SurfaceInteraction * si, void* mat_data)
{

}

extern "C" __device__ Vec3f __continuation_callable__brdf_disney(SurfaceInteraction* si, void* mat_data, const Vec3f& p)
{
    const auto* disney = (Disney::Data*)mat_data;
    si->emission = Vec3f(0.0f);
    
    const Vec3f V = -normalize(si->wo);
    const Vec3f L = normalize(p - si->p);
    const Vec3f N = si->shading.n;

    const float NdotV = fabs(dot(N, V));
    const float NdotL = fabs(dot(N, L));

    if (NdotV == 0.0f || NdotL == 0.0f)
        return Vec3f(0.0f);

    const Vec3f H = normalize(V + L);
    const float NdotH = dot(N, H);
    const float LdotH = dot(L, H);

    const Vec3f base = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        disney->base.prg_id, si, disney->base.data);
    si->albedo = base;

    // Diffuse term
    const float Fd90 = 0.5f + 2.0f * disney->roughness * LdotH * LdotH;
    const float FVd90 = fresnelSchlickT(NdotV, Fd90);
    const float FLd90 = fresnelSchlickT(NdotL, Fd90);
    const Vec3f f_diffuse = (base * math::inv_pi) * FVd90 * FLd90;

    // Subsurface
    const float Fss90 = disney->roughness * LdotH * LdotH;
    const float FVss90 = fresnelSchlickT(NdotV, Fss90);
    const float FLss90 = fresnelSchlickT(NdotL, Fss90);
    const Vec3f f_subsurface = (base * math::inv_pi) * 1.25f * (FVss90 * FLss90 * ((1.0f / (NdotV * NdotL)) - 0.5f) + 0.5f);

    // Sheen
    const Vec3f rho_tint = base / luminance(base);
    const Vec3f rho_sheen = lerp(Vec3f(1.0f), rho_tint, disney->sheen_tint);
    const Vec3f f_sheen = disney->sheen * rho_sheen * powf(1.0f - LdotH, 5.0f);

    // Specular
    const Vec3f X = si->shading.dpdu;
    const Vec3f Y = si->shading.dpdv;
    const float alpha = fmaxf(0.001f, disney->roughness);
    const float aspect = sqrtf(1.0f - disney->anisotropic * 0.9f);
    const float ax = fmaxf(0.001f, pow2(alpha) / aspect);
    const float ay = fmaxf(0.001f, pow2(alpha) * aspect);
    const Vec3f rho_specular = lerp(Vec3f(1.0f), rho_tint, disney->specular_tint);
    const Vec3f Fs0 = lerp(0.08f * disney->specular * rho_specular, base, disney->metallic);
    const Vec3f FHs0 = fresnelSchlickR(LdotH, Fs0);
    const float Ds = GTR2_aniso(NdotH, dot(H, X), dot(H, Y), ax, ay);
    float Gs = smithG_GGX_aniso(NdotL, dot(L, X), dot(L, Y), ax, ay);
    Gs *= smithG_GGX_aniso(NdotV, dot(V, X), dot(V, Y), ax, ay);
    const Vec3f f_specular = FHs0 * Fs0 * Gs;

    // Clearcoat
    const float Fcc = fresnelSchlickR(LdotH, 0.04f);
    const float alpha_cc = lerp(0.1f, 0.001f, disney->clearcoat_gloss);
    const float Dcc = GTR1(NdotH, alpha_cc);
    const float Gcc = smithG_GGX(NdotV, 0.25f);
    const Vec3f f_clearcoat = Vec3f(0.25f * disney->clearcoat * (Fcc * Dcc * Gcc));

    const Vec3f out = (1.0f - disney->metallic) * (lerp(f_diffuse, f_subsurface, disney->subsurface) + f_sheen) + f_specular + f_clearcoat;
    return out * fmaxf(NdotL, 0.0f);
}

extern "C" __device__ float __direct_callable__pdf_disney(SurfaceInteraction* si, void* mat_data, const Vec3f& p)
{

}

extern "C" __device__ Vec3f __direct_callable__area_emitter(SurfaceInteraction* si, void* surface_data)
{
    const auto* area = reinterpret_cast<AreaEmitter::Data*>(surface_data);
    si->trace_terminate = true;
    float is_emitted = dot(si->wo, si->shading.n) < 0.0f ? 1.0f : 0.0f;
    if (area->twosided)
    {
        is_emitted = 1.0f;
        si->shading.n = faceforward(si->shading.n, -si->wi, si->shading.n);
    }

    const Vec3f base = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
        area->texture.prg_id, si, area->texture.data);
    si->albedo = base;
    
    si->emission = base * area->intensity * is_emitted;
    return si->emission;
}

// Textures ----------------------------------------------------------------------
extern "C" __device__ Vec3f __direct_callable__bitmap(SurfaceInteraction* si, void* tex_data)
{
    const auto* bitmap = (BitmapTexture::Data*)tex_data;
    float4 c = tex2D<float4>(bitmap->texture, si->shading.uv.x(), si->shading.uv.y());
    return Vec3f(c);
}

extern "C" __device__ Vec3f __direct_callable__constant(SurfaceInteraction* si, void* tex_data)
{
    const auto* constant = (ConstantTexture::Data*)tex_data;
    return constant->color;
}

extern "C" __device__ Vec3f __direct_callable__checker(SurfaceInteraction* si, void* tex_data)
{
    const auto* checker = (CheckerTexture::Data*)tex_data;
    const Vec2f uv = si->shading.uv;
    const bool is_odd = sinf(uv.x() * math::pi * checker->scale) * sinf(uv.y() * math::pi * checker->scale) < 0.0f;
    return lerp(checker->color1, checker->color2, (float)is_odd);
}