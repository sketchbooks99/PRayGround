#include "hip/hip_runtime.h"
#include <prayground/prayground.h>
#include "params.h"

extern "C" { __constant__ LaunchParams params; }

using SurfaceInteraction = SurfaceInteraction_<Vec3f>;

struct LightInteraction {
    /* Surface point on the light source */
    Vec3f p;
    /* Surface normal on the light source */
    Vec3f n;
    /* Texture coordinate on light source */
    Vec2f uv;
    /* Area of light source */
    float area;
    /* PDF of light source */
    float pdf;
    /* Emission from light */
    Vec3f emission;
};

struct BSDFSample {
    Vec3f value;
    float pdf;
    Vec3f wi;
};

struct ScatteredRay {
    Vec3f reflected;
    Vec3f transmitted;
    float reflect_prob;
    // 1: Reflected, 2: Transmitted, 1 | 2: Both
    uint8_t scattered_type; 
};

static INLINE DEVICE void trace(
    OptixTraversableHandle handle,
    const Vec3f& ro,
    const Vec3f& rd,
    const float tmin,
    const float tmax,
    SurfaceInteraction* si
) 
{
    uint32_t u0, u1;
    packPointer(si, u0, u1);
    optixTrace(handle, ro, rd, tmin, tmax, 0.0f, 
        OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE, 
        0, 2, 0, u0, u1);
}

static INLINE DEVICE bool traceShadowRay(
    OptixTraversableHandle handle,
    const Vec3f& ro, 
    const Vec3f& rd,
    const float tmin,
    const float tmax
) 
{
    uint32_t hit = 0u;
    optixTrace(handle, ro, rd, tmin, tmax, 0.0f, 
        OptixVisibilityMask(1), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT, 
        1, 2, 1, hit);
    return (bool)hit;
}

static INLINE DEVICE float balanceHeuristic(float pdf1, float pdf2) {
    return pdf1 / (pdf1 + pdf2);
}

static INLINE DEVICE float powerHeuristic(float pdf1, float pdf2) {
    const float p1 = pdf1 * pdf1;
    const float p2 = pdf2 * pdf2;
    return p1 / (p1 + p2);
}

// ----------------------------------------------------------------------------
// Ray generation
// ----------------------------------------------------------------------------
extern "C" DEVICE void __raygen__pinhole() {
    const pgRaygenData<Camera>* rg = reinterpret_cast<pgRaygenData<Camera>*>(optixGetSbtDataPointer());

    const int frame = params.frame;

    const Vec3ui idx(optixGetLaunchIndex());

    const int image_idx = idx.y() * params.width + idx.x();
    uint32_t seed = tea<4>(image_idx, frame);

    Vec3f result(0.0f);

    int i = params.samples_per_launch;

    while (i > 0) {
        const Vec2f jitter = UniformSampler::get2D(seed) - 0.5f;
        const Vec2f d = 2.0f * Vec2f(
            (float)idx.x() + jitter.x(),
            (float)idx.y() + jitter.y()
        ) / Vec2f(params.width, params.height) - 1.0f;

        Vec3f ro, rd;
        getCameraRay(rg->camera, d.x(), d.y(), ro, rd);

        Vec3f throughput(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = 0.0f;
        si.albedo = 0.0f;
        si.trace_terminate = false;
        SurfaceInfo surface_info;
        surface_info.type == SurfaceType::None;
        si.surface_info = &surface_info;

        int depth = 0;
        for (;;) {
            if (depth >= params.max_depth)
                break;

            trace(params.handle, ro, rd, 1e-3f, 1e10f, &si);

            if (si.trace_terminate) {
                result += throughput * si.emission;
                break;
            }


            if (si.surface_info->type == SurfaceType::AreaEmitter) {
                // Evaluating emission from emitter
                Vec3f emission = optixDirectCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info->callable_id.sample, &si, si.surface_info->data
                );

                result += throughput * emission;
                if (si.trace_terminate)
                    break;
            } 
            // Specular surfaces
            else if (+(si.surface_info->type & SurfaceType::Delta)) {
                // Sample scattered ray
                auto wi = optixDirectCall<ScatteredRay, SurfaceInteraction*, void*>(
                    si.surface_info->callable_id.sample, &si, si.surface_info->data
                );
                // Both
                if (wi.scattered_type & 3)
                    si.wi = rnd(si.seed) < wi.reflect_prob ? wi.reflected : wi.transmitted;
                else if (wi.scattered_type & 1)
                    si.wi = wi.reflected;
                else if (wi.scattered_type & 2)
                    si.wi = wi.transmitted;

                // Evaluate BSDF
                Vec3f bsdf = optixDirectCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&, const Vec3f&>(
                    si.surface_info->callable_id.bsdf, &si, si.surface_info->data, si.wi, si.wo
                );

                throughput *= bsdf;
            }
            // Rough surface sampling with MIS
            else if (+(si.surface_info->type & SurfaceType::Rough)) {
                auto wi = optixDirectCall<ScatteredRay, SurfaceInteraction*, void*>(
                    si.surface_info->callable_id.sample, &si, si.surface_info->data
                );
                si.wi = wi.scattered_type & 1 ? wi.reflected : wi.transmitted;

                Vec3f bsdf = optixDirectCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&, const Vec3f&>(
                    si.surface_info->callable_id.bsdf, &si, si.surface_info->data, si.wi, si.wo
                );

                float pdf = optixDirectCall<float, SurfaceInteraction*, void*, const Vec3f&, const Vec3f&>(
                    si.surface_info->callable_id.pdf, &si, si.surface_info->data, si.wi, si.wo
                );

                throughput *= bsdf / pdf;
                //LightInfo light;
                //if (params.num_lights > 0) {
                //    const int light_id = rndInt(si.seed, 0, params.num_lights - 1);
                //    light = params.lights[light_id];
                //}

                //float pdf = 0.0f;

                //if (params.num_lights > 0) {
                //    LightInteraction li;
                //    // Sampling light point
                //    optixDirectCall<void, SurfaceInteraction*, LightInfo*, LightInteraction*, void*>(
                //        si.surface_info->callable_id.sample, &si, &light, &li, si.surface_info->data
                //    );
                //    Vec3f to_light = li.p - si.p;
                //    const float dist = length(to_light);
                //    const Vec3f light_dir = normalize(to_light);

                //    // For light PDF
                //    {
                //        const float t_shadow = dist_to_light - 1e-3f;
                //        // Trace shadow ray
                //        const bool is_hit = traceShadowRay(
                //            params.handle, si.p, light_dir, 1e-3f, t_shadow);

                //        // Next event estimation
                //        if (!hit_object) {
                //            const Vec3f bsdf = optixDirectCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
                //                si.surface_info->callable_id.bsdf, &si, si.surface_info->data, light_dir);

                //            const float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*, const Vec3f&>(
                //                si.surface_info->callable_id.pdf, &si, si.surface_info->data, light_dir);

                //            const float cos_theta = dot(-light_dir, li.n);

                //            // MIS weight
                //            const float weight = balanceHeuristic(li.pdf, bsdf_pdf * cos_theta / dist);

                //            result += weight * li.emission * bsdf * throughput / li.pdf;
                //        }
                //    }

                //    // Evaluate BSDFSample
                //    {
                //        // Importance sampling according to the BSDFSample
                //        optixDirectCall<void, SurfaceInteraction*, void*>(
                //            si.surface_info->callable_id.sample, &si, si.surface_info->data);
                //        
                //        // Evaluate BSDFSample
                //        const Vec3f bsdf = optixDirectCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&>(
                //            si.surface_info->callable_id.bsdf, &si, si.surface_info->data, si.wi);

                //        float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*, const Vec3f&>(
                //            si.surface_info->callable_id.pdf, &si, si.surface_info->data, si.wi);

                //        const float light_pdf = optixDirectCall<float, const LightInfo&, const Vec3f&, const Vec3f&, LightInteraction&>(
                //            light.pdf_id, light, si.p, light_dir, li);
                //        
                //        const float weight = balanceHeuristic(bsdf_pdf, light_pdf);
                //        throughput *= weight * bsdf / bsdf_pdf;
                //    }
                //}
            }

            ro = si.p;
            rd = si.wi;

            ++depth;
        } // for (;;)
        i--;
    } // while (i > 0)

    if (!result.isValid()) result = 0.0f;

    Vec3f accum_color = result / static_cast<float>(params.samples_per_launch);

    if (frame > 0) {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const Vec3f accum_color_prev = params.accum_buffer[image_idx];
        accum_color = lerp(accum_color_prev, accum_color, a);
    }

    params.accum_buffer[image_idx] = Vec4f(accum_color, 1.0f);
    Vec3u color = make_color(accum_color);
    params.result_buffer[image_idx] = Vec4u(color, 255);
}

// ----------------------------------------------------------------------------
// Miss program
// ----------------------------------------------------------------------------
extern "C" DEVICE void __miss__envmap() {
    pgMissData* data = reinterpret_cast<pgMissData*>(optixGetSbtDataPointer());
    auto* env = reinterpret_cast<EnvironmentEmitter::Data*>(data->env_data);
    SurfaceInteraction* si = getPtrFromTwoPayloads<SurfaceInteraction, 0>();

    Ray ray = getWorldRay();

    Shading shading;
    float t;
    const Sphere::Data env_sphere{ Vec3f(0.0f), 1e8f };
    pgIntersectionSphere(&env_sphere, ray, &shading, &t);

    si->shading.uv = shading.uv;
    si->trace_terminate = true;
    si->emission = optixDirectCall<Vec3f, const Vec2f&, void*>(env->texture.prg_id, si->shading.uv, env->texture.data);
}

extern "C" DEVICE void __miss__shadow() {
    setPayload<0>(0u);
}

// ----------------------------------------------------------------------------
// Light sampling
// ----------------------------------------------------------------------------
// Plane light sampling
extern "C" DEVICE void __direct_callable__sample_light_plane(
    const LightInfo& light, 
    const Vec3f& p, 
    LightInteraction& li, 
    uint32_t& seed)
{
    const auto* plane = (const Plane::Data*)light.shape_data;

    // Sample local point on the area emitter
    const float x = rnd(seed, plane->min.x(), plane->max.x());
    const float z = rnd(seed, plane->min.y(), plane->max.y());

    Vec3f rnd_p(x, 0.0f, z);
    rnd_p = light.objToWorld.pointMul(rnd_p);
    li.p = rnd_p;
    li.n = normalize(light.objToWorld.normalMul(Vec3f(0.0f, 1.0f, 0.0f)));
    li.uv = Vec2f(
        (x - plane->min.x()) / (plane->max.x() - plane->min.x()), 
        (z - plane->min.y()) / (plane->max.y() - plane->min.y()));
    
    // Calcluate area of the light source
    const Vec3f p0 = light.objToWorld.pointMul(Vec3f(plane->min.x(), 0.0f, plane->min.y()));
    const Vec3f p1 = light.objToWorld.pointMul(Vec3f(plane->max.x(), 0.0f, plane->min.y()));
    const Vec3f p2 = light.objToWorld.pointMul(Vec3f(plane->min.x(), 0.0f, plane->max.y()));
    li.area = length(cross(p1 - p0, p2 - p0));

    const Vec3f wi = rnd_p - p;
    const float t = length(wi);
    const float cos_theta = fabs(dot(li.n, normalize(wi)));
    if (cos_theta < math::eps)
        li.pdf = 0.0f;
    else
        li.pdf = t * t / (li.area * cos_theta);

    // Emission from light source
    const auto* area_light = (const AreaEmitter::Data*)light.surface_info->data;
    float is_emitted = 1.0f;
    if (!area_light->twosided)
        is_emitted = (float)(dot(li.n, normalize(wi)) > 0.0f);
    const Vec3f base = optixDirectCall<Vec3f, const Vec2f&, void*>(
        area_light->texture.prg_id, li.uv, area_light->texture.data);
    li.emission = is_emitted * base * area_light->intensity;
}

// Triangle light sampling
static INLINE DEVICE Vec3f randomSampleOnTriangle(uint32_t& seed, const Triangle& triangle) {

    Vec2f uv = UniformSampler::get2D(seed);

    return barycentricInterop(triangle.v0, triangle.v1, triangle.v2, uv);
}

extern "C" DEVICE void __direct_callable__sample_light_triangle(
    const LightInfo& light,
    const Vec3f& p,
    LightInteraction& li,
    uint32_t& seed)
{
    const auto* triangle = (const Triangle*)light.shape_data;

    // Sample local point on the light
    const Vec2f uv = UniformSampler::get2D(seed);
    li.p = randomSampleOnTriangle(seed, *triangle);
    li.n = normalize(triangle->n);
    li.uv = uv;
    li.area = 0.5f * length(cross(triangle->v1 - triangle->v0, triangle->v2 - triangle->v0));

    // PDF
    const Vec3f wi = li.p - p;
    Vec3f N = triangle->n;
    N = faceforward(N, -wi, N);
    const float t = length(wi);
    const float cos_theta = fabs(dot(N, normalize(wi)));
    if (cos_theta < math::eps)
        li.pdf = 0.0f;
    else
        li.pdf = t * t / (li.area * cos_theta);

    // Emission from light source
    const auto* area_light = (const AreaEmitter::Data*)light.surface_info->data;
    float is_emitted = 1.0f;
    if (!area_light->twosided)
        is_emitted = (float)(dot(li.n, normalize(wi)) > 0.0f);
    const Vec3f base = optixDirectCall<Vec3f, const Vec2f&, void*>(
        area_light->texture.prg_id, li.uv, area_light->texture.data);
    li.emission = is_emitted * base * area_light->intensity;
}

// ----------------------------------------------------------------------------
// Hitgroups
// ----------------------------------------------------------------------------
// Sphere
extern "C" DEVICE void __intersection__sphere() {
    const pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());
    const Sphere::Data* sphere = reinterpret_cast<Sphere::Data*>(data->shape_data);

    Ray ray = getLocalRay();
    pgReportIntersectionSphere(sphere, ray);
}

extern "C" DEVICE void __intersection__plane() {
    const pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());
    const Plane::Data* plane = reinterpret_cast<Plane::Data*>(data->shape_data);

    Ray ray = getLocalRay();
    pgReportIntersectionPlane(plane, ray);
}

extern "C" DEVICE void __closesthit__custom() {
    const pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());

    Ray ray = getWorldRay();

    Shading* shading = getPtrFromTwoAttributes<Shading, 0>();

    //  Transform shading from object to world space
    shading->n = normalize(optixTransformNormalFromObjectToWorldSpace(shading->n));
    shading->dpdu = normalize(optixTransformVectorFromObjectToWorldSpace(shading->dpdu));
    shading->dpdv = normalize(optixTransformVectorFromObjectToWorldSpace(shading->dpdv));

    auto* si = getPtrFromTwoPayloads<SurfaceInteraction, 0>();

    si->p = ray.at(ray.tmax);
    si->shading = *shading;
    si->t = ray.tmax;
    si->wo = -ray.d;
    si->surface_info = data->surface_info;
}

// Mesh
extern "C" DEVICE void __closesthit__mesh() {
    const pgHitgroupData* data = reinterpret_cast<pgHitgroupData*>(optixGetSbtDataPointer());
    const TriangleMesh::Data* mesh_data = reinterpret_cast<TriangleMesh::Data*>(data->shape_data);

    Ray ray = getWorldRay();

    Shading shading = pgGetMeshShading(mesh_data, optixGetTriangleBarycentrics(), optixGetPrimitiveIndex());

    SurfaceInteraction* si = getPtrFromTwoPayloads<SurfaceInteraction, 0>();


    if (data->surface_info->use_bumpmap) {
        Frame shading_frame = Frame::FromXZ(shading.dpdu, shading.n);
        // Fetch bumpmap normal 
        Vec3f n = optixDirectCall<Vec3f, Vec2f&, void*>(data->surface_info->bumpmap.prg_id, shading.uv, data->surface_info->bumpmap.data);
        n = normalize(n * 2.0f - 1.0f);
        // Transform normal from tangent space to local space
        n = shading_frame.fromLocal(n);
        shading.n = normalize(n);
    }

    // Transform shading from object to world space
    shading.n = normalize(optixTransformNormalFromObjectToWorldSpace(shading.n));
    shading.dpdu = optixTransformVectorFromObjectToWorldSpace(shading.dpdu);
    shading.dpdv = optixTransformVectorFromObjectToWorldSpace(shading.dpdv);

    si->p = ray.at(ray.tmax);
    si->shading = shading;
    si->t = ray.tmax;
    si->wo = -ray.d;
    si->surface_info = data->surface_info;
}

extern "C" DEVICE void __closesthit__shadow() {
    setPayload<0>(1u);
}

// ----------------------------------------------------------------------------
// Surface 
// ----------------------------------------------------------------------------
// Diffuse
extern "C" DEVICE ScatteredRay __direct_callable__sample_diffuse(SurfaceInteraction* si, void* data, const Vec3f& wo) {
    const Diffuse::Data* diffuse = reinterpret_cast<Diffuse::Data*>(data);

    si->wi = pgImportanceSamplingDiffuse(diffuse, si->wo, si->shading, si->seed);
    si->trace_terminate = false;

    return { si->wi, Vec3f(0.0f), 1.0f, 1};
}

extern "C" DEVICE Vec3f __direct_callable__bsdf_diffuse(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Diffuse::Data* diffuse = reinterpret_cast<Diffuse::Data*>(data);

    const Vec3f albedo = optixDirectCall<Vec3f, const Vec2f&, void*>(
        diffuse->texture.prg_id, si->shading.uv, diffuse->texture.data);
    si->albedo = albedo;
    si->emission = 0.0f;
    return albedo * pgGetDiffuseBRDF(wi, si->shading.n);
}

extern "C" DEVICE float __direct_callable__pdf_diffuse(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    return pgGetDiffusePDF(wi, si->shading.n);
}

// Specular reflection
extern "C" DEVICE ScatteredRay __direct_callable__sample_conductor(SurfaceInteraction* si, void* data, const Vec3f& wo) {
    const Conductor::Data* conductor = reinterpret_cast<Conductor::Data*>(data);

    if (conductor->twosided)
        si->shading.n = faceforward(si->shading.n, si->wo, si->shading.n);
    si->wi = reflect(-si->wo, si->shading.n);
    si->trace_terminate = false;
    return { si->wi, Vec3f(0.0f), 1.0f, 1};
}

extern "C" DEVICE Vec3f __direct_callable__bsdf_conductor(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Conductor::Data* conductor = reinterpret_cast<Conductor::Data*>(data);

    const Vec3f albedo = optixDirectCall<Vec3f, const Vec2f&, void*>(
        conductor->texture.prg_id, si->shading.uv, conductor->texture.data);
    si->albedo = albedo;
    // Apply thinfilm interaction
    const float cos_theta = dot(si->wo, si->shading.n);
    Vec3f tf_thickness = optixDirectCall<Vec3f, const Vec2f&, void*>(conductor->thinfilm.thickness.prg_id, si->shading.uv, conductor->thinfilm.thickness.data);
    tf_thickness *= conductor->thinfilm.thickness_scale;
    Vec3f thinfilm = fresnelAiry(1.0f, cos_theta, conductor->thinfilm.ior, conductor->thinfilm.extinction, tf_thickness.x(), conductor->thinfilm.tf_ior);

    return thinfilm * albedo;
}

extern "C" float __direct_callable__pdf_conductor(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    return 1.0f;
}

// Specular transmission
extern "C" DEVICE ScatteredRay __direct_callable__sample_dielectric(SurfaceInteraction* si, void* data, const Vec3f& wo) {
    const Dielectric::Data* dielectric = reinterpret_cast<Dielectric::Data*>(data);

    float ni = 1.000292f;       /// @todo Consider IOR of current medium where ray goes on
    float nt = dielectric->ior;
    Vec3f wo = -si->wo;
    float cosine = dot(wo, si->shading.n);
    // Check where the ray is going outside or inside
    bool into = cosine < 0;
    Vec3f outward_normal = into ? si->shading.n : -si->shading.n;

    // Swap IOR based on ray location
    if (!into) swap(ni, nt);

    // Check if the ray can be refracted
    cosine = fabs(cosine);
    float sine = sqrtf(1.0f - pow2(cosine));
    bool cannot_refract = (ni / nt) * sine > 1.0f;

    // Get reflectivity by the Fresnel equation
    float reflect_prob = fresnel(cosine, ni, nt);
    // Get out going direction of the ray
    if (cannot_refract)
        return { reflect(wo, outward_normal), Vec3f(0.0f), 1.0f, 1 };
    else
        return { reflect(wo, outward_normal), refract(wo, outward_normal, cosine, ni, nt), reflect_prob, 1 | 2 };
}

extern "C" DEVICE Vec3f __direct_callable__bsdf_dielectric(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Dielectric::Data* dielectric = reinterpret_cast<Dielectric::Data*>(data);

    bool into = dot(wo, si->shading.n) > 0.0f;

    // Evaluate BSDFSample
    const Vec3f albedo = optixDirectCall<Vec3f, const Vec2f&, void*>(
        dielectric->texture.prg_id, si->shading.uv, dielectric->texture.data);
    si->albedo = albedo;
    si->emission = 0.0f;
    const float cos_theta = dot(wo, si->shading.n);
    float ni = 1.0f;
    float nt = dielectric->ior;
    if (!into)
        swap(ni, nt);

    float sin_theta = sqrtf(fmaxf(0.0f, 1.0f - cos_theta * cos_theta));
    bool cannot_refract = (ni / nt) * sin_theta > 1.0f;

    Vec3f tf_thickness = optixDirectCall<Vec3f, const Vec2f&, void*>(dielectric->thinfilm.thickness.prg_id, si->shading.uv, dielectric->thinfilm.thickness.data);
    Vec3f tf_value = fresnelAiry(1.0f, cos_theta, dielectric->ior, dielectric->thinfilm.extinction, tf_thickness.x(), dielectric->thinfilm.tf_ior) * albedo;

    Vec3f bsdf = albedo;
    if (into)
        bsdf *= tf_value;
    return bsdf;
}

extern "C" DEVICE float __direct_callable__pdf_dielectric(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    return 1.0f;
}

// Disney
extern "C" DEVICE ScatteredRay __direct_callable__sample_disney(SurfaceInteraction* si, void* data, const Vec3f& wo) {
    const Disney::Data* disney = reinterpret_cast<Disney::Data*>(data);

    // Importance sampling
    si->wi = pgImportanceSamplingDisney(disney, -si->wo, si->shading, si->seed);
    si->trace_terminate = false;
    return { si->wi, Vec3f(0.0f), 1.0f, 1 };
}

extern "C" DEVICE Vec3f __direct_callable__bsdf_disney(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Disney::Data* disney = reinterpret_cast<Disney::Data*>(data);

    // Evaluate BSDF
    const Vec3f albedo = optixDirectCall<Vec3f, const Vec2f&, void*>(disney->albedo.prg_id, si->shading.uv, disney->albedo.data);
    si->albedo = albedo;
    const float cos_theta = dot(si->wi, si->shading.n);
    Vec3f tf_thickness = optixDirectCall<Vec3f, const Vec2f&, void*>(disney->thinfilm.thickness.prg_id, si->shading.uv, disney->thinfilm.thickness.data);
    tf_thickness *= disney->thinfilm.thickness_scale;

    Vec3f tf_value = fresnelAiry(1.0f, cos_theta, disney->thinfilm.ior, disney->thinfilm.extinction, tf_thickness.x(), disney->thinfilm.tf_ior);
    float mag_albedo = length(albedo);
    tf_value = normalize(tf_value) * mag_albedo;
    Vec3f bsdf = pgGetDisneyBRDF(disney, -si->wo, si->wi, si->shading, tf_value);
    return bsdf;
}

extern "C" DEVICE float __direct_callable__pdf_disney(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Disney::Data* disney = reinterpret_cast<Disney::Data*>(data);

    // Evaluate PDF
    return pgGetDisneyPDF(disney, -si->wo, wi, si->shading);
}

// Layered material
extern "C" DEVICE ScatteredRay __direct_callable__sample_layered(SurfaceInteraction* si, void* data, const Vec3f& wo) {
    const Layered::Data* layered = reinterpret_cast<Layered::Data*>(data);

    uint32_t n_layers = layered->num_layers;
    int32_t l = 0;

    Vec3f wi;
    Vec3f _wo = wo;
    float sample_prob;
    int32_t l = 0;

    // Downward ray tracing into the layered surface
    while (l < n_layers) {
        SurfaceInfo info = si->surface_info[l];
        // Generate refraction and reflection ray and increment layer if interacting surface is refractive
        if (+(info.type & SurfaceType::Refractive)) {
            ScatteredRay scattered = optixDirectCall<ScatteredRay, SurfaceInteraction*, void*>(info.callable_id.sample, si, info.data);
            // If refractive surface cannot refract, generate reflection ray and don't increment layer
            if (rnd(si->seed) < scattered.reflect_prob)  {
                wi = scattered.reflected;
                break;
            }
            // Transmit the ray to the next layer
            else {
                _wo = -scattered.transmitted;
                ++l;
            }
        }
        // Generate reflection ray and don't increment layer
        else {
            ScatteredRay scattered = optixDirectCall<ScatteredRay, SurfaceInteraction*, void*>(info.callable_id.sample, si, info.data);
            wi = scattered.reflected;
            break;
        }
    }

    // Upward ray tracing from the bottom layer to the top layer
    for (int32_t i = l - 1; i >= 0; --i) {
        SurfaceInfo info = si->surface_info[i];
        ScatteredRay scattered = optixDirectCall<ScatteredRay, SurfaceInteraction*, void*>(
            info.callable_id.bsdf, si, info.data, wi);

        // Ray cannot go through from the surface to the outside
        if (rnd(si->seed) < scattered.reflect_prob) {
            si->trace_terminate = true;
        }
        else {
            wi = scattered.transmitted;
        }
    }

    return { wi, Vec3f(0.0f), 1.0f, 1 };
}

extern "C" DEVICE Vec3f __direct_callable__bsdf_layered(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Layered::Data* layered = reinterpret_cast<Layered::Data*>(data);

    uint32_t n_layers = layered->num_layers;

    int32_t l = 0;
    Vec3f total_bsdf;
    while (l < n_layers) {
        SurfaceInfo info = si->surface_info[l];
        Vec3f bsdf = optixDirectCall<Vec3f, SurfaceInteraction*, void*, const Vec3f&, const Vec3f&>(
            info.callable_id.bsdf, si, info.data, wi, wo);
        ++l;
    }
}

extern "C" DEVICE float __direct_callable__pdf_layered(SurfaceInteraction* si, void* data, const Vec3f& wi, const Vec3f& wo) {
    const Layered::Data* layered = reinterpret_cast<Layered::Data*>(data);

    uint32_t n_layers = layered->num_layers;
}

// Area emitter
extern "C" DEVICE Vec3f __direct_callable__area_emitter(SurfaceInteraction* si, void* data) {
    const AreaEmitter::Data* area = reinterpret_cast<AreaEmitter::Data*>(data);

    si->trace_terminate = true;
    float is_emitted = dot(si->wo, si->shading.n) < 0.0f ? 1.0f : 0.0f;
    if (area->twosided) {
        is_emitted = 1.0f;

        si->shading.n = faceforward(si->shading.n, si->wo, si->shading.n);
    }

    const Vec3f base = optixDirectCall<Vec3f, const Vec2f&, void*>(
        area->texture.prg_id, si->shading.uv, area->texture.data);
    si->albedo = base;
    si->emission = base * area->intensity * is_emitted;
    
    return si->emission;
}

// Textures
extern "C" DEVICE Vec3f __direct_callable__bitmap(const Vec2f& uv, void* data) {
    return pgGetBitmapTextureValue<Vec3f>(uv, data);
}

extern "C" DEVICE Vec3f __direct_callable__constant(const Vec2f& uv, void* data) {
    return pgGetConstantTextureValue<Vec3f>(uv, data);
}

extern "C" DEVICE Vec3f __direct_callable__checker(const Vec2f& uv, void* data) {
    return pgGetCheckerTextureValue<Vec3f>(uv, data);
}