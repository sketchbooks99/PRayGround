#include "hip/hip_runtime.h"
#include "util.cuh"

using namespace prayground;

static __forceinline__ __device__ void getCameraRay(
    const LensCamera::Data& camera, const float x, const float y, Vec3f& ro, Vec3f& rd
)
{
    rd = normalize(x * camera.U + y * camera.V + camera.W);
    ro = camera.origin;
}

static __forceinline__ __device__ Vec3f reinhardToneMap(const Vec3f& color, const float white)
{
    const float l = luminance(color);
    return (color * 1.0f) / (1.0f + l / white);
}

static __forceinline__ __device__ Vec3f exposureToneMap(const Vec3f& color, const float exposure)
{
    return Vec3f(1.0f) - expf(-color * exposure);
}

extern "C" __device__ void __raygen__pinhole()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const int frame = params.frame;
    const Vec3ui idx(optixGetLaunchIndex());
    unsigned int seed = tea<4>(idx.y() * params.width + idx.x(), frame);

    Vec3f result(0.0f);
    Vec3f normal(0.0f);

    int i = params.samples_per_launch;

    do
    {
        const Vec2f subpixel_jitter = UniformSampler::get2D(seed) - 0.5f;

        const Vec2f d = 2.0f * Vec2f(
            (static_cast<float>(idx.x()) + subpixel_jitter.x()) / params.width,
            (static_cast<float>(idx.y()) + subpixel_jitter.y()) / params.height
        ) - 1.0f;

        Vec3f ro, rd;
        getCameraRay(raygen->camera, d.x(), d.y(), ro, rd);

        Vec3f throughput(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = Vec3f(0.0f);
        si.albedo = Vec3f(0.0f);
        si.trace_terminate = false;

        float tmax = raygen->camera.farclip / dot(rd, normalize(raygen->camera.lookat - ro));

        int depth = 0;
        for ( ;; ) {

            if ( depth >= params.max_depth )
				break;

            trace(params.handle, ro, rd, 0.01f, tmax, 0, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            // Get emission from area emitter
            if ( si.surface_info.type == SurfaceType::AreaEmitter )
            {
                // Evaluating emission from emitter
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                result += si.emission * throughput;

                if (si.trace_terminate)
                    break;
            }
            // Specular sampling
            else if (+(si.surface_info.type & SurfaceType::Delta))
            {
                // Sampling scattered direction
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample , &si, si.surface_info.data);
                
                // Evaluate bsdf
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                throughput *= bsdf_val;
            }
            // Rough surface sampling with applying MIS
            else if ( +(si.surface_info.type & (SurfaceType::Rough | SurfaceType::Diffuse)) )
            {
                // Importance sampling according to the BSDF
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);

                // Evaluate PDF depends on BSDF
                float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.pdf, &si, si.surface_info.data);

                // Evaluate BSDF
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);

                throughput *= bsdf_val / bsdf_pdf;
            }

            // Make tmax large except for when the primary ray
            tmax = 1e16f;
            
            ro = si.p;
            rd = si.wi;

            if (depth == 0)
                normal = si.shading.n;

            ++depth;
        }
    } while (--i);

    const unsigned int image_index = idx.y() * params.width + idx.x();

    if (!result.isValid()) result = Vec3f(0.0f);

    Vec3f accum_color = result / static_cast<float>(params.samples_per_launch);

    if (frame > 0)
    {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const Vec3f accum_color_prev(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = Vec4f(accum_color, 1.0f);
    Vec3u color = make_color(reinhardToneMap(accum_color, params.white));
    params.result_buffer[image_index] = Vec4u(color, 255);
}

extern "C" __device__ void __raygen__lens()
{
    const RaygenData* raygen = reinterpret_cast<RaygenData*>(optixGetSbtDataPointer());

    const int frame = params.frame;
    const Vec3ui idx(optixGetLaunchIndex());
    unsigned int seed = tea<4>(idx.y() * params.width + idx.x(), frame);

    Vec3f result(0.0f);
    Vec3f normal(0.0f);

    int i = params.samples_per_launch;

    do
    {
        const Vec2f subpixel_jitter = UniformSampler::get2D(seed) - 0.5f;

        const Vec2f res(params.width, params.height);
        const Vec2f d = 2.0f * ((Vec2f(idx.x(), idx.y()) + subpixel_jitter) / res) - 1.0f;

        Vec3f ro, rd;
        getLensCameraRay(raygen->camera, d.x(), d.y(), ro, rd, seed);

        Vec3f throughput(1.0f);

        SurfaceInteraction si;
        si.seed = seed;
        si.emission = Vec3f(0.0f);
        si.albedo = Vec3f(0.0f);
        si.trace_terminate = false;

        float tmax = raygen->camera.farclip / dot(rd, normalize(raygen->camera.lookat - ro));

        int depth = 0;
        for ( ;; ) {

            if ( depth >= params.max_depth )
				break;

            trace(params.handle, ro, rd, 0.01f, tmax, 0, &si);

            if (si.trace_terminate) {
                result += si.emission * throughput;
                break;
            }

            // Get emission from area emitter
            if ( si.surface_info.type == SurfaceType::AreaEmitter )
            {
                // Evaluating emission from emitter
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                result += si.emission * throughput;

                if (si.trace_terminate)
                    break;
            }
            // Specular sampling
            else if (+(si.surface_info.type & SurfaceType::Delta))
            {
                // Sampling scattered direction
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);
                
                // Evaluate bsdf
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);
                throughput *= bsdf_val;
            }
            // Rough surface sampling with applying MIS
            else if ( +(si.surface_info.type & (SurfaceType::Rough | SurfaceType::Diffuse)) )
            {
                // Importance sampling according to the BSDF
                optixDirectCall<void, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.sample, &si, si.surface_info.data);

                // Evaluate PDF depends on BSDF
                float bsdf_pdf = optixDirectCall<float, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.pdf, &si, si.surface_info.data);

                // Evaluate BSDF
                Vec3f bsdf_val = optixContinuationCall<Vec3f, SurfaceInteraction*, void*>(
                    si.surface_info.callable_id.bsdf, &si, si.surface_info.data);

                throughput *= bsdf_val / bsdf_pdf;
            }

            // Make tmax large except for when the primary ray
            tmax = 1e16f;
            
            ro = si.p;
            rd = si.wi;

            if (depth == 0)
                normal = si.shading.n;

            ++depth;
        }
    } while (--i);

    const unsigned int image_index = idx.y() * params.width + idx.x();

    if (result.x() != result.x()) result.x() = 0.0f;
    if (result.y() != result.y()) result.y() = 0.0f;
    if (result.z() != result.z()) result.z() = 0.0f;

    Vec3f accum_color = result / static_cast<float>(params.samples_per_launch);

    if (frame > 0)
    {
        const float a = 1.0f / static_cast<float>(frame + 1);
        const Vec3f accum_color_prev(params.accum_buffer[image_index]);
        accum_color = lerp(accum_color_prev, accum_color, a);
    }
    params.accum_buffer[image_index] = Vec4f(accum_color, 1.0f);
    Vec3u color = make_color(reinhardToneMap(accum_color, params.white));
    params.result_buffer[image_index] = Vec4u(color, 255);
}

